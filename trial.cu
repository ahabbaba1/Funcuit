
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

typedef struct Node_Struct
{
	char op;
	char c1;
	char c2;
	char c3;
	char c4;
	char c5;
	int totChildren;

}Node;


//GPU Function
void gpu_cuit(int start, int end, int index, char* input) //end is exclusive
{
	char* c = (char*) malloc(5);
	strncpy(c, input+start, end-start+1);

printf("%d, end %d, index %d, c %s\n", start, end, index, c);
}
/*
///////////////////////////////////////////////////////////////////////////////////////////////////////////
// HELPER CODE TO INITIALIZE, PRINT AND TIME
struct timeval start, end;
void initialize(float *a, long N) {
  long i;
  for (i = 0; i < N; ++i) { 
    a[i] = pow(rand() % 10, 2); 
  }                                                                                                                                                                                       
}

void print(float* a, long N) {
   if (doPrint) {
   long i;
   for (i = 0; i < N; ++i)
      printf("%d ", (int) a[i]);
   printf("\n");
   }
}  

void starttime() {
  gettimeofday( &start, 0 );
}

void endtime(const char* c) {
   gettimeofday( &end, 0 );
   double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
   printf("%s: %f ms\n", c, elapsed); 
}

void init(float* a, long N, const char* c) {
  printf("***************** %s **********************\n", c);
  printf("Initializing array....\n");
  initialize(a, N); 
  printf("Done.\n");
  print(a, N);
  printf("Running %s...\n", c);
  starttime();
}

void finish(float* a, long N, const char* c) {
  endtime(c);
  printf("Done.\n");
  print(a, N);
  printf("***************************************************\n");
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////

*/

int main() //a is the array, N is the size of the array
{
	long N = 1024;
		
	//read in String from file

	FILE* file = fopen("data.txt","r");
	if(file == NULL)
	{
		printf("Could not open %s  make sure your file is valid.\n","data.txt");
		exit(0);
	}

	char input[500];
	int op[100];

	fgets(input, 100, file);

	printf("%s",input);
	//loop through String and populate indeces array
	int i = 0;
	int index = 0;

	char output[500];
	int j = 0;

	int start = 0;
	int end = 0;

	for(i = 0; i < strlen(input);i++)
	{
		if(input[i] == '+')
		{
			op[index] = i;
			index++;
			printf("Found a + at %d\n",i);
			output[j] = input[i];

		}

	}


	// Test: GPU


  // How many threads, how many cores?
  int numThreads = 1024; // This can vary, up to 1024
  long numCores = N / 1024 + 1;

int count = 0;
	for(i = 0; i <= index - 1;i++)
	{
		if(i == 0)
		{
			gpu_cuit(0, op[i] - 1, count, input);
		}else
		{
			gpu_cuit(op[i-1] + 1, op[i] - 1, count, input);
		}
		count++;	
	}


  

	exit(0);
}

