
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define SIZE 100
#define filename "data.txt"

typedef struct Node_Struct
{
	char op;
	char c1;
	char c2;
	char c3;
	char c4;
	char c5;
	int totChildren;

}Node;

//create array of type node, of size x;

//GPU Function
//Sents relevant arguements to GPU child which makes a node in position i of array
void gpu_cuit(int start, int end, int index, char* input) //end is exclusive
{
	char* c = (char*) malloc(5);
	strncpy(c, input+start, end-start+1);
	//create a Node
	//assign position i to Node
	printf("%d, end %d, index %d, c %s\n", start, end, index, c);
}

/*
///////////////////////////////////////////////////////////////////////////////////////////////////////////
// HELPER CODE TO INITIALIZE, PRINT AND TIME
struct timeval start, end;
void initialize(float *a, long N) {
long i;
for (i = 0; i < N; ++i) {
a[i] = pow(rand() % 10, 2);
}
}

void print(float* a, long N) {
if (doPrint) {
long i;
for (i = 0; i < N; ++i)
printf("%d ", (int) a[i]);
printf("\n");
}
}

void starttime() {
gettimeofday( &start, 0 );
}

void endtime(const char* c) {
gettimeofday( &end, 0 );
double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
printf("%s: %f ms\n", c, elapsed);
}

void init(float* a, long N, const char* c) {
printf("***************** %s **********************\n", c);
printf("Initializing array....\n");
initialize(a, N);
printf("Done.\n");
print(a, N);
printf("Running %s...\n", c);
starttime();
}

void finish(float* a, long N, const char* c) {
endtime(c);
printf("Done.\n");
print(a, N);
printf("***************************************************\n");
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////

*/

int main()
{

	long N = 1024;
	char input[SIZE * 5 + SIZE]; 	//maximum number of arguments + operators
	int op[SIZE];      //holds indexes / indices of operators of the input
	int i = 0;		     //loop variable
	int index = 0;     //holds the position of the last element of the op array
	int start = 0;
	int end = 0;

	//define file object and read it in
	FILE* file = fopen(filename,"r");
	if(file == NULL)
	{
		printf("Could not open %s. Please make sure your file is valid.\n", filename);
		exit(0);
	}

	//read in equation
	fgets(input, 100, file);

	//iterate through and save operator indexes
	for(i = 0; i < strlen(input);i++)
	{
		if(input[i] == '+')
		{
			op[index] = i;
			index++;
		}
	}

	// Test: GPU

	// How many threads, how many cores?
	int numThreads = 1024; // This can vary, up to 1024
	long numCores = N / 1024 + 1;

	int count = 0;

	//send to gpu children
	for(i = 0; i <= index - 1;i++)
	{
		if(i == 0)
		{
			gpu_cuit(0, op[i] - 1, count, input);
		}else
		{
			gpu_cuit(op[i-1] + 1, op[i] - 1, count, input);
		}
		count++;
	}

	exit(0);
}
