
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define SIZE 100
#define filename "data.txt"

typedef struct Node_Struct
{
	char op;
	char c1;
	char c2;
	char c3;
	char c4;
	char c5;
} Node;

//create array of type node, of size x;
Node* nodes = (Node*)malloc((sizeof(Node) * SIZE));

//GPU Function
//Sents relevant arguements to GPU child which makes a node in position i of array

__global__ void gpu_cuit(int totalPlusses, int* op, char* input) //end is exclusive
{
	//char* lclvar = "actual string";
	//printf("top of gpu method %s\n", lclvar);
	int i = threadIdx.x;
	int start;
	int end;
	int j;

	//printf("IN GPU: gpu paramaters: int totalPlusses: (%d), int* op (%p), %s, op[%d]: %d \n", totalPlusses, op, input, i, op[i]);

	if(i == 0)
	{
		//printf("inside if i == 0, op[%d]: %d\n", i,op[i]);
		start = 0;
		end = op[i] - 1;
	}
	else
	{
		//printf("inside if i != 0, op[%d]: %d\n", i, op[i]);
		start = op[i - 1] + 1;
		end  = op[i] - 1;
	}

	printf("assignmet of start (%d) and end (%d). totalOps: %d THREAD: %d\n", start, end, totalPlusses, i);

}

	//create a Node
//	Node* temp = (Node*)malloc(sizeof(Node));
//temp->op = '+';
	//int l = 0;
	/*
	while(substring[l] != NULL)
	{
		printf("infiniteeeeeeeeeee");
		if (l == 0)
			temp->c1 = substring[l];
		else if (l == 1)
			temp->c2 = substring[l];
		else if (l == 2)
			temp->c3 = substring[l];
		else if (l == 3)
			temp->c4 = substring[l];
		else if (l == 4)
			temp->c5 = substring[l];
	}*/

	//assign position i to Node
//
//		printf("currArrIndex %d, %s\n", currArrIndex, substring);


/*
///////////////////////////////////////////////////////////////////////////////////////////////////////////
// HELPER CODE TO INITIALIZE, PRINT AND TIME
struct timeval start, end;
void initialize(float *a, long N) {
long i;
for (i = 0; i < N; ++i) {
a[i] = pow(rand() % 10, 2);
}
}

void print(float* a, long N) {
if (doPrint) {
long i;
for (i = 0; i < N; ++i)
printf("%d ", (int) a[i]);
printf("\n");
}
}

void starttime() {
gettimeofday( &start, 0 );
}

void endtime(const char* c) {
gettimeofday( &end, 0 );
double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
printf("%s: %f ms\n", c, elapsed);
}

void init(float* a, long N, const char* c) {
printf("***************** %s **********************\n", c);
printf("Initializing array....\n");
initialize(a, N);
printf("Done.\n");
print(a, N);
printf("Running %s...\n", c);
starttime();
}

void finish(float* a, long N, const char* c) {
endtime(c);
printf("Done.\n");
print(a, N);
printf("***************************************************\n");
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////

*/

int main()
{
	char input[SIZE * 5 + SIZE]; //= (char*)malloc(sizeof(char) * (SIZE * 5 + SIZE)); 	//maximum number of arguments + operators      //holds indexes / indices of operators of the input
	int i = 0;		     //loop variable
	int totalPlusses = 0;     //holds the position of the last element of the op array
	int currArrIndex = 0;
	//long N = 64000000;

	//define file object and read it in
	FILE* file = fopen(filename,"r");
	if(file == NULL)
	{
		printf("Could not open %s. Please make sure your file is valid.\n", filename);
		exit(0);
	}

	//read in equation
	fgets(input, SIZE, file);

	int* op; //= (int*) malloc(sizeof(int) * SIZE);
	hipMallocManaged(&op, SIZE);

	int* gpu_op;
	char* gpu_input;
	hipMalloc(&gpu_op, sizeof(int) * SIZE);
	hipMalloc(&gpu_input, sizeof(char) * (SIZE * 5 + SIZE));

	//iterate through and save operator indexes
	for(i = 0; i < strlen(input); i++)
	{
		if(input[i] == '+')
		{
			op[totalPlusses] = i;
			totalPlusses++;
		}
	}

	hipMemcpy(gpu_op, op, (sizeof(int) * SIZE), hipMemcpyDeviceToHost);
	hipMemcpy(gpu_input, input,  sizeof(char) * (SIZE * 5 + SIZE), hipMemcpyDeviceToHost);

	char* sub;
	hipMallocManaged(&sub, 1000);
	strcpy(sub, input);

	printf("before gpu call\n");
	printf("IN MAIN: gpu paramaters: int totalPlusses: (%d), int* op (%p), char* input (%p) \n", totalPlusses, op, sub);

	for (i = 0; i < totalPlusses; i++)
	{
		printf("op[%d]: %d \n", i, op[i]);
	}

	gpu_cuit<<<1, 8>>>(totalPlusses, op, sub);
	printf("after gpu call\n");

	hipDeviceSynchronize();

	printf("after gpu synchronize\n");

	//free(op);
	//sfree(input);
	hipFree(gpu_op);
	hipFree(gpu_input);

	exit(0);
}
