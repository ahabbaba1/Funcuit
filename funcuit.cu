
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#define SIZE 10000
#define filename "data.txt"

typedef struct Node_Struct
{
	char op;
	char c1;
	char c2;
	char c3;
	char c4;
	char c5;
} Node;


//GPU Function
//Sents relevant arguements to GPU child which makes a node in position i of array
__global__ void gpu_cuit(int totalPlusses, int* op, char* input, Node** nodes) //end is exclusive
{
	int i = threadIdx.x;
	int start;
	int end;
	int j;

	if(i == 0)
	{
		start = 0;
		end = op[i] - 1;
	}
	else
	{
		start = op[i - 1] + 1;
		end  = op[i] - 1;
	}

	Node* temp = (Node*)malloc(sizeof(Node));
	temp->op = '+';
	int l = 0;

	for(j = start; j <= end; j++ )
	{
		if (input[j] == NULL)
			break;
		else
			if (l == 0)
				{temp->c1 = input[j]; l++;}
			else if (l == 1)
				{temp->c2 = input[j]; l++;}
			else if (l == 2)
				{temp->c3 = input[j]; l++;}
			else if (l == 3)
				{temp->c4 = input[j]; l++;}
			else if (l == 4)
				{temp->c5 = input[j]; l++;}
	}

	nodes[i] = temp;
}

void normal_cuit(int totalPlusses, int* op, char* input, Node** nodes)
{
	int start;
	int end;
	int i;

	for (i = 0; i < totalPlusses; i++)
	{
		if(i == 0)
		{
			start = 0;
			end = op[i] - 1;
		}
		else
		{
			start = op[i - 1] + 1;
			end  = op[i] - 1;
		}

		Node* temp = (Node*)malloc(sizeof(Node));
		temp->op = '+';
		int l = 0;
		int j = 0;
		for(j = start; j <= end; j++ )
		{
				if (l == 0)
					{temp->c1 = input[j]; l++;}
				else if (l == 1)
					{temp->c2 = input[j]; l++;}
				else if (l == 2)
					{temp->c3 = input[j]; l++;}
				else if (l == 3)
					{temp->c4 = input[j]; l++;}
				else if (l == 4)
					{temp->c5 = input[j]; l++;}
		}
		nodes[i] = temp;
	}
}

struct timeval start, end;

void starttime() {
gettimeofday( &start, 0 );
}

void endtime() {
gettimeofday( &end, 0 );
double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
printf("Time: %f ms\n",elapsed);
}

int main()
{
	char* input;
	hipMallocManaged(&input, sizeof(char) * SIZE * 5 + SIZE);
	int i = 0;		//loop variable
	int totalPlusses = 0;     //holds the position of the last element of the op array
	int* op; 						//holds indexes / indices of operators of the input
	hipMallocManaged(&op, SIZE);
	Node** nodes_gpu;
	hipMallocManaged(&nodes_gpu, sizeof(Node*) * SIZE);
	Node** nodes_normal;
	hipMallocManaged(&nodes_normal, sizeof(Node*) * SIZE);

	//define file object and read it in
	FILE* file = fopen(filename,"r");
	if(file == NULL)
	{
		printf("Could not open %s. Please make sure your file is valid.\n", filename);
		exit(0);
	}

	//read in equation
	fgets(input, SIZE, file);

	//iterate through and save operator indexes
	for(i = 0; i < strlen(input); i++)
	{
		if(input[i] == '+')
		{
			op[totalPlusses] = i;
			totalPlusses++;
		}
	}

	//gpu timed test
	starttime();
	gpu_cuit<<<1, totalPlusses>>>(totalPlusses, op, input, nodes_gpu);
	hipDeviceSynchronize();
	endtime();

	//iterative timed test
	starttime();
	normal_cuit(totalPlusses, op, input, nodes_normal);
	endtime();


	//free everything
	hipFree(input);
	for (i = 0; i < totalPlusses; i++)
	{
		free(nodes_gpu[i]);
		free(nodes_normal[i]);
	}
	hipFree(op);
	hipFree(nodes_gpu);
	hipFree(nodes_normal);

	exit(0);
}
