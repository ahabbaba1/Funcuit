
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

long N = 64000000;  // Play with this value                                                                                                                                                                                        
int doPrint = 0; 

////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GPU CODE
//
// Normal C function to square root values
void normal(float* a, long N)                                                                                                                                                                                     
{
  long i;                                                                                                                                                                                                                
  for (i = 0; i < N; ++i)                                                                                                                                                                                    
    a[i] = sqrt(a[i]);                                                                                                                                                                                           
}                 

// GPU function to square root values
__global__ void gpu_sqrt(float* a, long N) {
   long element = blockIdx.x*blockDim.x + threadIdx.x; // Each thread must get a different element
   if (element < N) a[element] = sqrt(a[element]);
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////
                                                                                                                                                                                               

///////////////////////////////////////////////////////////////////////////////////////////////////////////
// HELPER CODE TO INITIALIZE, PRINT AND TIME
struct timeval start, end;
void initialize(float *a, long N) {
  long i;
  for (i = 0; i < N; ++i) { 
    a[i] = pow(rand() % 10, 2); 
  }                                                                                                                                                                                       
}

void print(float* a, long N) {
   if (doPrint) {
   long i;
   for (i = 0; i < N; ++i)
      printf("%d ", (int) a[i]);
   printf("\n");
   }
}  

void starttime() {
  gettimeofday( &start, 0 );
}

void endtime(const char* c) {
   gettimeofday( &end, 0 );
   double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
   printf("%s: %f ms\n", c, elapsed); 
}

void init(float* a, long N, const char* c) {
  printf("***************** %s **********************\n", c);
  printf("Initializing array....\n");
  initialize(a, N); 
  printf("Done.\n");
  print(a, N);
  printf("Running %s...\n", c);
  starttime();
}

void finish(float* a, long N, const char* c) {
  endtime(c);
  printf("Done.\n");
  print(a, N);
  printf("***************************************************\n");
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////



int main()                                                                                                                                                                                  
{
  float* a = (float*) malloc(N*sizeof(float));
  ///////////////////////////////////////////////
  // Test 1: Sequential For Loop
  init(a, N, "Normal");
  normal(a, N); 
  finish(a, N, "Normal"); 
  ///////////////////////////////////////////////

  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  // Test 2: GPU
  init(a, N, "GPU");

  // How many threads, how many cores?
  int numThreads = 1024; // This can vary, up to 1024
  long numCores = N / 1024 + 1;

  float* gpuA;

  hipMalloc(&gpuA, N*sizeof(float)); // 1. Allocate enough memory on the GPU
  hipMemcpy(gpuA, a, N*sizeof(float), hipMemcpyHostToDevice); // 2. Copy original array from CPU to GPU
  gpu_sqrt<<<numCores, numThreads>>>(gpuA, N);  // 3. Each GPU thread square roots its value
  hipMemcpy(a, gpuA, N*sizeof(float), hipMemcpyDeviceToHost); // 4. Copy square rooted values from GPU to CPU
  hipFree(&gpuA); // 5. Free the memory on the GPU


  finish(a, N, "GPU");
  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  free(a);
  return 0;
}

