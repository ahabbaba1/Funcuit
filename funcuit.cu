
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define SIZE 100
#define filename "data.txt"

typedef struct Node_Struct
{
	char op;
	char c1;
	char c2;
	char c3;
	char c4;
	char c5;
} Node;

//create array of type node, of size x;
Node* nodes = (Node*)malloc((sizeof(Node) * SIZE));

//GPU Function
//Sents relevant arguements to GPU child which makes a node in position i of array

__global__ void gpu_cuit(int totalOps, int* op, char* input) //end is exclusive
{
	printf("top of gpu method\n");
	int i = threadIdx.x;
	int start;
	int end;
	if(i == 0)
	{
		printf("inside if i == 0, op[%d]: %d\n", i,op[i]);
		start = 0;
		end = op[i] - 1;
	}
	else
	{
			printf("inside if i != 0, op[%d]: %d\n", i,op[i]);
		start = op[i - 1] + 1;
		end  = op[i] - 1;
	}

	printf("assignmet of start (%d) and end (%d). totalOps: %d THREAD: %d\n", start, end, totalOps, i);
		int j;
		for(j = start; j <= end; j++)
		{
			printf("Our substring: %c", input[j]);
		}


	//create a Node
//	Node* temp = (Node*)malloc(sizeof(Node));
//temp->op = '+';
	//int l = 0;
	/*
	while(substring[l] != NULL)
	{
		printf("infiniteeeeeeeeeee");
		if (l == 0)
			temp->c1 = substring[l];
		else if (l == 1)
			temp->c2 = substring[l];
		else if (l == 2)
			temp->c3 = substring[l];
		else if (l == 3)
			temp->c4 = substring[l];
		else if (l == 4)
			temp->c5 = substring[l];
	}*/

	//assign position i to Node
//
//		printf("currArrIndex %d, %s\n", currArrIndex, substring);
}

/*
///////////////////////////////////////////////////////////////////////////////////////////////////////////
// HELPER CODE TO INITIALIZE, PRINT AND TIME
struct timeval start, end;
void initialize(float *a, long N) {
long i;
for (i = 0; i < N; ++i) {
a[i] = pow(rand() % 10, 2);
}
}

void print(float* a, long N) {
if (doPrint) {
long i;
for (i = 0; i < N; ++i)
printf("%d ", (int) a[i]);
printf("\n");
}
}

void starttime() {
gettimeofday( &start, 0 );
}

void endtime(const char* c) {
gettimeofday( &end, 0 );
double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
printf("%s: %f ms\n", c, elapsed);
}

void init(float* a, long N, const char* c) {
printf("***************** %s **********************\n", c);
printf("Initializing array....\n");
initialize(a, N);
printf("Done.\n");
print(a, N);
printf("Running %s...\n", c);
starttime();
}

void finish(float* a, long N, const char* c) {
endtime(c);
printf("Done.\n");
print(a, N);
printf("***************************************************\n");
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////

*/

int main()
{
	char input[SIZE * 5 + SIZE]; 	//maximum number of arguments + operators
	int op[SIZE];      //holds indexes / indices of operators of the input
	int i = 0;		     //loop variable
	int totalOps = 0;     //holds the position of the last element of the op array
	int currArrIndex = 0;
	long N = 64000000;

	//define file object and read it in
	FILE* file = fopen(filename,"r");
	if(file == NULL)
	{
		printf("Could not open %s. Please make sure your file is valid.\n", filename);
		exit(0);
	}

	//read in equation
	fgets(input, SIZE, file);

	printf("%d \n", i );

	//op = (int*) malloc(sizeof(int) * SIZE);

	int newVar = 0;
	//iterate through and save operator indexes
	for(i = 0; i < strlen(input); i++)
	{

		if(input[i] == '+')
		{
			printf("i: %d \n", i );


			op[newVar] = i;

			newVar = newVar + 1;
			printf("op[%d]: %d \n", newVar, op[newVar] );
		}
	}

	// Test: GPU
	//cudaMallocManaged(&sub, 6);

	long numCores = N / 1024 + 1;
	int numThreads = 1024;

	printf("before gpu call\n");
//	gpu_cuit<<<1, 8>>>(totalOps, op, input);
	printf("after gpu call\n");

	hipDeviceSynchronize();

	printf("after gpu synchronize\n");

	exit(0);
}
