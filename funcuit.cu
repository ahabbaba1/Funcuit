
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define SIZE 100
#define filename "data.txt"

typedef struct Node_Struct
{
	char op;
	char c1;
	char c2;
	char c3;
	char c4;
	char c5;
	int totChildren;

}Node;

//create array of type node, of size x;


//GPU Function
//Sents relevant arguements to GPU child which makes a node in position i of array
__global__ void gpu_cuit(int count, int index, char* input) //end is exclusive
{

	int i = threadIdx.x;
	//create a Node
	//assign position i to Node
	if(i < index)
	printf("count %d, %s\n", count, input);
}

/*
///////////////////////////////////////////////////////////////////////////////////////////////////////////
// HELPER CODE TO INITIALIZE, PRINT AND TIME
struct timeval start, end;
void initialize(float *a, long N) {
long i;
for (i = 0; i < N; ++i) {
a[i] = pow(rand() % 10, 2);
}
}

void print(float* a, long N) {
if (doPrint) {
long i;
for (i = 0; i < N; ++i)
printf("%d ", (int) a[i]);
printf("\n");
}
}

void starttime() {
gettimeofday( &start, 0 );
}

void endtime(const char* c) {
gettimeofday( &end, 0 );
double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
printf("%s: %f ms\n", c, elapsed);
}

void init(float* a, long N, const char* c) {
printf("***************** %s **********************\n", c);
printf("Initializing array....\n");
initialize(a, N);
printf("Done.\n");
print(a, N);
printf("Running %s...\n", c);
starttime();
}

void finish(float* a, long N, const char* c) {
endtime(c);
printf("Done.\n");
print(a, N);
printf("***************************************************\n");
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////

*/

int main()
{

	
	char input[SIZE * 5 + SIZE]; 	//maximum number of arguments + operators
	long N = sizeof(input);
	int op[SIZE];      //holds indexes / indices of operators of the input
	int i = 0;		     //loop variable
	int index = 0;     //holds the position of the last element of the op array
	int start = 0;
	int end = 0;

	//define file object and read it in
	FILE* file = fopen(filename,"r");
	if(file == NULL)
	{
		printf("Could not open %s. Please make sure your file is valid.\n", filename);
		exit(0);
	}

	//read in equation
	fgets(input, 100, file);

	//iterate through and save operator indexes
	for(i = 0; i < strlen(input);i++)
	{
		if(input[i] == '+')
		{
			op[index] = i;
			index++;
		}
	}

	// Test: GPU

	// How many threads, how many cores?
	int numThreads = 1024; // This can vary, up to 1024
	long numCores = N / 1024 + 1;

	int count = 0;

	char* sub;
	hipMallocManaged(&sub, 5);
//	strcpy(sub, input);

	//send to gpu children
	for(i = 0; i <= index - 1;i++)
	{

		if(i == 0)
		{
			strncpy(sub, input, op[i]);
			gpu_cuit<<<1,1>>>(count, index, sub);
		}else
		{
			strncpy(sub, input+op[i-1] + 1, op[i]-op[i-1]-1);
			gpu_cuit<<<1,1>>>(count, index, sub);
		}

		hipDeviceSynchronize();
		count++;
	}

	exit(0);
}
