
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#define SIZE 10000
#define BLOCKSIZE 1
#define filename "data.txt"

typedef struct Node_Struct
{
	char op;
	char c1;
	char c2;
	char c3;
	char c4;
	char c5;
} Node;

//GPU Function
//Sents relevant arguements to GPU child which makes a node in position i of array
__global__ void gpu_cuit_all_shared(int totalPlusses, int* op, char* input, Node** nodes) //end is exclusive
{
	int i = threadIdx.x;
	__shared__ int op_shared[BLOCKSIZE];
	__shared__ char input_shared[BLOCKSIZE];

	int start;
	int end;
	int j;

	op_shared[i] = op[i];
	input_shared[i] = input[i];

	if(i == 0)
	{
		start = 0;
		end = op_shared[i] - 1;
	}
	else
	{
		start = op_shared[i - 1] + 1;
		end  = op_shared[i] - 1;
	}

	Node* temp = (Node*)malloc(sizeof(Node));
	temp->op = '+';
	int l = 0;

	for(j = start; j <= end; j++ )
	{
		if (input_shared[j] == NULL)
			break;
		else
			if (l == 0)
				{temp->c1 = input_shared[j]; l++;}
			else if (l == 1)
				{temp->c2 = input_shared[j]; l++;}
			else if (l == 2)
				{temp->c3 = input_shared[j]; l++;}
			else if (l == 3)
				{temp->c4 = input_shared[j]; l++;}
			else if (l == 4)
				{temp->c5 = input_shared[j]; l++;}
	}

	nodes[i] = temp;
}

__global__ void gpu_cuit_op_shared(int totalPlusses, int* op, char* input, Node** nodes) //end is exclusive
{
	int i = threadIdx.x;
	__shared__ int op_shared[BLOCKSIZE];

	int start;
	int end;
	int j;

	op_shared[i] = op[i];

	if(i == 0)
	{
		start = 0;
		end = op_shared[i] - 1;
	}
	else
	{
		start = op_shared[i - 1] + 1;
		end  = op_shared[i] - 1;
	}

	Node* temp = (Node*)malloc(sizeof(Node));
	temp->op = '+';
	int l = 0;

	for(j = start; j <= end; j++ )
	{
		if (input[j] == NULL)
			break;
		else
			if (l == 0)
				{temp->c1 = input[j]; l++;}
			else if (l == 1)
				{temp->c2 = input[j]; l++;}
			else if (l == 2)
				{temp->c3 = input[j]; l++;}
			else if (l == 3)
				{temp->c4 = input[j]; l++;}
			else if (l == 4)
				{temp->c5 = input[j]; l++;}
	}

	nodes[i] = temp;
}

__global__ void gpu_cuit_input_shared(int totalPlusses, int* op, char* input, Node** nodes) //end is exclusive
{
	int i = threadIdx.x;
	__shared__ char input_shared[BLOCKSIZE];

	int start;
	int end;
	int j;

	input_shared[i] = input[i];

	if(i == 0)
	{
		start = 0;
		end = op[i] - 1;
	}
	else
	{
		start = op[i - 1] + 1;
		end  = op[i] - 1;
	}

	Node* temp = (Node*)malloc(sizeof(Node));
	temp->op = '+';
	int l = 0;

	for(j = start; j <= end; j++ )
	{
		if (input_shared[j] == NULL)
			break;
		else
			if (l == 0)
				{temp->c1 = input_shared[j]; l++;}
			else if (l == 1)
				{temp->c2 = input_shared[j]; l++;}
			else if (l == 2)
				{temp->c3 = input_shared[j]; l++;}
			else if (l == 3)
				{temp->c4 = input_shared[j]; l++;}
			else if (l == 4)
				{temp->c5 = input_shared[j]; l++;}
	}

	nodes[i] = temp;
}

//Sents relevant arguements to GPU child which makes a node in position i of array
__global__ void gpu_cuit_none_shared(int totalPlusses, int* op, char* input, Node** nodes) //end is exclusive
{
	int i = threadIdx.x;
	int start;
	int end;
	int j;

	op[i] = op[i];
	input[i] = input[i];

	if(i == 0)
	{
		start = 0;
		end = op[i] - 1;
	}
	else
	{
		start = op[i - 1] + 1;
		end  = op[i] - 1;
	}

	Node* temp = (Node*)malloc(sizeof(Node));
	temp->op = '+';
	int l = 0;

	for(j = start; j <= end; j++ )
	{
		if (input[j] == NULL)
			break;
		else
			if (l == 0)
				{temp->c1 = input[j]; l++;}
			else if (l == 1)
				{temp->c2 = input[j]; l++;}
			else if (l == 2)
				{temp->c3 = input[j]; l++;}
			else if (l == 3)
				{temp->c4 = input[j]; l++;}
			else if (l == 4)
				{temp->c5 = input[j]; l++;}
	}

	nodes[i] = temp;
}

void normal_cuit(int totalPlusses, int* op, char* input, Node** nodes)
{
	int start;
	int end;
	int i;

	for (i = 0; i < totalPlusses; i++)
	{
		if(i == 0)
		{
			start = 0;
			end = op[i] - 1;
		}
		else
		{
			start = op[i - 1] + 1;
			end  = op[i] - 1;
		}

		Node* temp = (Node*)malloc(sizeof(Node));
		temp->op = '+';
		int l = 0;
		int j = 0;
		for(j = start; j <= end; j++ )
		{
				if (l == 0)
					{temp->c1 = input[j]; l++;}
				else if (l == 1)
					{temp->c2 = input[j]; l++;}
				else if (l == 2)
					{temp->c3 = input[j]; l++;}
				else if (l == 3)
					{temp->c4 = input[j]; l++;}
				else if (l == 4)
					{temp->c5 = input[j]; l++;}
		}
		nodes[i] = temp;
	}
}

struct timeval start, end;

void starttime()
{
	gettimeofday( &start, 0 );
}

double endtime()
{
	gettimeofday( &end, 0 );
	return (( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0);
}

void simulate(int iterations, int totalPlusses, int* op, char* input, Node** nodes, int version)
{
	int i = 0;
	double avg = 0.0;
	while(i < iterations)
	{
		starttime();

		if(version == 0)
			gpu_cuit_all_shared<<<1, totalPlusses>>>(totalPlusses, op, input, nodes);
		else if(version == 1)
			gpu_cuit_op_shared<<<1, totalPlusses>>>(totalPlusses, op, input, nodes);
		else if(version == 2)
			gpu_cuit_input_shared<<<1, totalPlusses>>>(totalPlusses, op, input, nodes);
		else if(version == 3)
			gpu_cuit_none_shared<<<1, totalPlusses>>>(totalPlusses, op, input, nodes);
		else if(version == 4)
			normal_cuit(totalPlusses, op, input, nodes);

		hipDeviceSynchronize();
		avg += endtime();
		i++;
	}

	if (version == 0)
		printf("GPU -  all shared arrays: ");
	else if (version == 1)
		printf("    -    op shared array: ");
	else if (version == 2)
		printf("    - input shared array: ");
	else if (version == 3)
		printf("    -   no shared arrays: ");
	else if (version == 4)
		printf("\nSequential computation: ");

	avg = avg / iterations;
	printf("%lf\n", avg);
}

int main()
{
	char* input;	//stores read in string
	hipMallocManaged(&input, sizeof(char) * SIZE * 5 + SIZE);
	int i = 0;		//loop variable
	int totalPlusses = 0;     //holds the position of the last element of the op array
	int* op; 						//holds indexes / indices of operators of the input
	hipMallocManaged(&op, SIZE);
	Node** nodes_gpu;
	hipMallocManaged(&nodes_gpu, sizeof(Node*) * SIZE);
	Node** nodes_normal;
	hipMallocManaged(&nodes_normal, sizeof(Node*) * SIZE);

	//define file object and read it in
	FILE* file = fopen(filename,"r");
	if(file == NULL)
	{
		printf("Could not open %s. Please make sure your file is valid.\n", filename);
		exit(0);
	}

	//read in equation
	fgets(input, SIZE, file);

	//iterate through and save operator indexes
	for(i = 0; i < strlen(input); i++)
	{
		if(input[i] == '+')
		{
			op[totalPlusses] = i;
			totalPlusses++;
		}
	}

	int n = 10000;

	printf("Average time (s) with %d iterations\n", n);
	//gpu timed tests
	//both arrays shared
	simulate(n, totalPlusses, op, input, nodes_gpu, 0);
	//op array shared
	simulate(n, totalPlusses, op, input, nodes_gpu, 1);
	//input array shared
	simulate(n, totalPlusses, op, input, nodes_gpu, 2);
	//no arrays shared
	simulate(n, totalPlusses, op, input, nodes_gpu, 3);

	//serial timed test
	simulate(n, totalPlusses, op, input, nodes_normal, 4);

	//free everything
	hipFree(input);
	for (i = 0; i < totalPlusses; i++)
	{
		free(nodes_gpu[i]);
		free(nodes_normal[i]);
	}
	hipFree(op);
	hipFree(nodes_gpu);
	hipFree(nodes_normal);

	exit(0);
}
