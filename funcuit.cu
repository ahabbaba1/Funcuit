/************************************************************************************
 * Converts a boolean equation in DNF form to a list of nodes.
 * 		Compares parallel computations with varied shared memory allocation
 *    and and iterative implementation.
 *
 * We affirm that we wrote this program ourselves in accordance to FIU the Code of
 * Academic Integrity.
 *    Authors: Alejandro Ravelo
 *             Alejandro Koszarycz
 *             Aysha Habbaba
 *             Rahul Mittal
 **********************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <getopt.h>
#define SIZE 10000                //max number of operations
#define BLOCKSIZE 1               //total blocks
#define filename "data.txt"       //filename is by default data.txt

//represents each 'subtree' with values op and children c1-c5
typedef struct Node_Struct
{
	char op, c1, c2, c3, c4, c5;
} Node;

//***GPU Functions***/

//Creates a node representing the operation at point i, and adds it to the nodes array at position i
//Both operator indicies array and input string are part of the block's share'd memory
__global__ void gpu_cuit_all_shared(int totalPlusses, char* op, int* opIndexes, char* input, Node** nodes) //end is exclusive
{
	int i = threadIdx.x;
	__shared__ char op_shared[BLOCKSIZE];
	__shared__ int opIndexes_shared[BLOCKSIZE];
	__shared__ char input_shared[BLOCKSIZE];

	int start;
	int end;
	int j;

	op_shared[i] = op[i];
	opIndexes_shared[i] = opIndexes[i];
	input_shared[i] = input[i];

	if(i == 0)
	{
		start = 0;
		end = opIndexes_shared[i] - 1;
	}
	else
	{
		start = opIndexes_shared[i - 1] + 1;
		end  = opIndexes_shared[i] - 1;
	}

	Node* temp = (Node*)malloc(sizeof(Node));
	temp->op = op_shared[i];
	temp->c1 = ' ';
	temp->c2 = ' ';
	temp->c3 = ' ';
	temp->c4 = ' ';
	temp->c5 = ' ';
	int l = 0;

	for(j = start; j <= end; j++ )
	{
		if (input_shared[j] == NULL)
			break;
		else
			if (l == 0)
				{temp->c1 = input_shared[j]; l++;}
			else if (l == 1)
				{temp->c2 = input_shared[j]; l++;}
			else if (l == 2)
				{temp->c3 = input_shared[j]; l++;}
			else if (l == 3)
				{temp->c4 = input_shared[j]; l++;}
			else if (l == 4)
				{temp->c5 = input_shared[j]; l++;}
	}

	nodes[i] = temp;
}

//Creates a node representing the operation at point i, and adds it to the nodes array at position i
//Operator indices array is part of the block's shared memory
__global__ void gpu_cuit_op_shared(int totalPlusses, char* op ,int* opIndexes, char* input, Node** nodes) //end is exclusive
{
	int i = threadIdx.x;
	__shared__ char op_shared[BLOCKSIZE];
	__shared__ int opIndexes_shared[BLOCKSIZE];

	int start;
	int end;
	int j;

	op_shared[i] = op[i];
	opIndexes_shared[i] = opIndexes[i];

	if(i == 0)
	{
		start = 0;
		end = opIndexes_shared[i] - 1;
	}
	else
	{
		start = opIndexes_shared[i - 1] + 1;
		end  = opIndexes_shared[i] - 1;
	}

	Node* temp = (Node*)malloc(sizeof(Node));
	temp->op = op_shared[i];
	temp->c1 = ' ';
	temp->c2 = ' ';
	temp->c3 = ' ';
	temp->c4 = ' ';
	temp->c5 = ' ';
	int l = 0;

	for(j = start; j <= end; j++ )
	{
		if (input[j] == NULL)
			break;
		else
			if (l == 0)
				{temp->c1 = input[j]; l++;}
			else if (l == 1)
				{temp->c2 = input[j]; l++;}
			else if (l == 2)
				{temp->c3 = input[j]; l++;}
			else if (l == 3)
				{temp->c4 = input[j]; l++;}
			else if (l == 4)
				{temp->c5 = input[j]; l++;}
	}

	nodes[i] = temp;
}

//Creates a node representing the operation at point i, and adds it to the nodes array at position i
//Input string is part of the block's shared memory
__global__ void gpu_cuit_input_shared(int totalPlusses, char* op, int* opIndexes, char* input, Node** nodes) //end is exclusive
{
	int i = threadIdx.x;
	__shared__ char input_shared[BLOCKSIZE];

	int start;
	int end;
	int j;

	input_shared[i] = input[i];

	if(i == 0)
	{
		start = 0;
		end = opIndexes[i] - 1;
	}
	else
	{
		start = opIndexes[i - 1] + 1;
		end  = opIndexes[i] - 1;
	}

	Node* temp = (Node*)malloc(sizeof(Node));
	temp->op = op[i];
	temp->c1 = ' ';
	temp->c2 = ' ';
	temp->c3 = ' ';
	temp->c4 = ' ';
	temp->c5 = ' ';
	int l = 0;

	for(j = start; j <= end; j++ )
	{
		if (input_shared[j] == NULL)
			break;
		else
			if (l == 0)
				{temp->c1 = input_shared[j]; l++;}
			else if (l == 1)
				{temp->c2 = input_shared[j]; l++;}
			else if (l == 2)
				{temp->c3 = input_shared[j]; l++;}
			else if (l == 3)
				{temp->c4 = input_shared[j]; l++;}
			else if (l == 4)
				{temp->c5 = input_shared[j]; l++;}
	}

	nodes[i] = temp;
}

//Creates a node representing the operation at point i, and adds it to the nodes array at position i
//no shared memory used
__global__ void gpu_cuit_none_shared(int totalPlusses, char* op, int* opIndexes, char* input, Node** nodes) //end is exclusive
{
	int i = threadIdx.x;
	int start;
	int end;
	int j;

	if(i == 0)
	{
		start = 0;
		end = opIndexes[i] - 1;
	}
	else
	{
		start = opIndexes[i - 1] + 1;
		end  = opIndexes[i] - 1;
	}

	Node* temp = (Node*)malloc(sizeof(Node));
	temp->op = op[i];
	temp->c1 = ' ';
	temp->c2 = ' ';
	temp->c3 = ' ';
	temp->c4 = ' ';
	temp->c5 = ' ';
	int l = 0;

	for(j = start; j <= end; j++ )
	{
		if (input[j] == NULL)
			break;
		else
			if (l == 0)
				{temp->c1 = input[j]; l++;}
			else if (l == 1)
				{temp->c2 = input[j]; l++;}
			else if (l == 2)
				{temp->c3 = input[j]; l++;}
			else if (l == 3)
				{temp->c4 = input[j]; l++;}
			else if (l == 4)
				{temp->c5 = input[j]; l++;}
	}

	nodes[i] = temp;
}

//***Iterative Function****/

//Creates a node representing the operation at point i, and adds it to the nodes array at position i
//Uses a for loop to replicate the above thread independent methods
void normal_cuit(int totalPlusses, char* op, int* opIndexes, char* input, Node** nodes)
{
	int start;
	int end;
	int i;

	for (i = 0; i < totalPlusses; i++)
	{
		if(i == 0)
		{
			start = 0;
			end = opIndexes[i] - 1;
		}
		else
		{
			start = opIndexes[i - 1] + 1;
			end  = opIndexes[i] - 1;
		}

		Node* temp = (Node*)malloc(sizeof(Node));
		temp->op = op[i];
		temp->c1 = ' ';
		temp->c2 = ' ';
		temp->c3 = ' ';
		temp->c4 = ' ';
		temp->c5 = ' ';
		int l = 0;
		int j = 0;
		for(j = start; j <= end; j++ )
		{
				if (l == 0)
					{temp->c1 = input[j]; l++;}
				else if (l == 1)
					{temp->c2 = input[j]; l++;}
				else if (l == 2)
					{temp->c3 = input[j]; l++;}
				else if (l == 3)
					{temp->c4 = input[j]; l++;}
				else if (l == 4)
					{temp->c5 = input[j]; l++;}
		}
		nodes[i] = temp;
	}
}

//***Utility Methods**/

//Timing
struct timeval start, end;

void start_time()
{
	gettimeofday( &start, 0 );
}

double end_time()
{
	gettimeofday( &end, 0 );
	return (( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0);
}

//print nodes list
void print_nodes(Node** nodes, int totalNodes)
{
	for (int i = 0; i < totalNodes; i++)
	{
		printf("Operation: %c\n\tOperands: ", nodes[i]->op);

		//print only relavant children
		if (nodes[i]->c1 != ' ' && nodes[i]->c2 == ' ')
			printf("c1: %c\n", nodes[i]->c1);
		else if (nodes[i]->c1 != ' ' && nodes[i]->c2 != ' ' && nodes[i]->c3 == ' ')
			printf("c1: %c, c2: %c\n", nodes[i]->c1, nodes[i]->c2);
		else if (nodes[i]->c1 != ' ' && nodes[i]->c2 != ' ' && nodes[i]->c3 != ' ' && nodes[i]->c4 == ' ')
			printf("c1: %c, c2: %c, c3: %c\n", nodes[i]->c1, nodes[i]->c2, nodes[i]->c3);
		else if (nodes[i]->c1 != ' ' && nodes[i]->c2 != ' ' && nodes[i]->c3 != ' ' && nodes[i]->c4 != ' ' && nodes[i]->c5 == ' ')
			printf("c1: %c, c2: %c, c3: %c, c4: %c\n", nodes[i]->c1, nodes[i]->c2, nodes[i]->c3, nodes[i]->c4);
		else
			printf("c1: %c, c2: %c, c3: %c, c4: %c, c5: %c\n", nodes[i]->c1, nodes[i]->c2, nodes[i]->c3, nodes[i]->c4, nodes[i]->c5);
	}
}

//simulates the running of the gpu and iterative cuit methods n times, and calcuates the average elapsed
//takes all necessary arguments for the cuit methods plus
	//int n - number of iterations
	//int version - which cuit method to call (0 = all_shared, 1 = op_shared, 2 = input_shared, 3 = none_shared, 4 = normal)
void simulate(int n, int totalPlusses, char* op, int* opIndexes, char* input, Node** nodes, int version)
{
	int i = 0;
	double avg = 0.0;
	while(i < n)
	{
		start_time();

		if(version == 0)
			gpu_cuit_all_shared<<<BLOCKSIZE, totalPlusses>>>(totalPlusses, op, opIndexes, input, nodes);
		else if(version == 1)
			gpu_cuit_op_shared<<<BLOCKSIZE, totalPlusses>>>(totalPlusses, op, opIndexes, input, nodes);
		else if(version == 2)
			gpu_cuit_input_shared<<<BLOCKSIZE, totalPlusses>>>(totalPlusses, op, opIndexes, input, nodes);
		else if(version == 3)
			gpu_cuit_none_shared<<<BLOCKSIZE, totalPlusses>>>(totalPlusses, op, opIndexes, input, nodes);
		else if(version == 4)
			normal_cuit(totalPlusses, op, opIndexes, input, nodes);

		hipDeviceSynchronize();
		avg += end_time();
		i++;
	}

	if (version == 0)
		printf("GPU -  all shared arrays: ");
	else if (version == 1)
		printf("    -    op shared array: ");
	else if (version == 2)
		printf("    - input shared array: ");
	else if (version == 3)
		printf("    -   no shared arrays: ");
	else if (version == 4)
		printf("\nSequential computation: ");

	avg = avg / n;
	printf("%lf\n", avg);
}

int main(int argc, char **argv)
{
	int c = 0;                   //getopt var
	int print = 0;               //if 1, print nodes
	char* input;                 //stores read in string
	hipMallocManaged(&input, sizeof(char) * SIZE * 5 + SIZE);
	int i = 0;                  //loop variable
	int totalPlusses = 0;       //holds the position of the last element of the op array
	int* opIndexes;             //holds indexes / indices of operators of the input
	hipMallocManaged(&opIndexes, sizeof(int) * SIZE);
	char* op;                   //holds operators of the input
	hipMallocManaged(&op, sizeof(char) * SIZE);
	Node** nodes_gpu;
	hipMallocManaged(&nodes_gpu, sizeof(Node*) * SIZE);
	Node** nodes_normal;
	hipMallocManaged(&nodes_normal, sizeof(Node*) * SIZE);

	//allows for command line command to enable node list printing
	while ((c = getopt(argc, argv, "p")) != -1) {
			 if (c == 'p')
			 	print = 1;
			 else
			 {
					 printf("Sorry, you entered an invalid flag. Please try again.");
					 exit(0);
					 break;
			 }
	 }

	//define file object and read it in
	FILE* file = fopen(filename,"r");
	if(file == NULL)
	{
		printf("Could not open %s. Please make sure your file is valid.\n", filename);
		exit(0);
	}

	//read in equation
	fgets(input, SIZE, file);

	//iterate through and save operator indexes
	for(i = 0; i < strlen(input); i++)
	{
		if(input[i] == '+' || input[i] == '*')
		{
			op[totalPlusses] = input[i];
			opIndexes[totalPlusses] = i;
			totalPlusses++;
		}
	}

	int n = 10000;

	printf("Average time (s) with %d iterations\n", n);
	//gpu timed tests
	//both arrays shared
	simulate(n, totalPlusses, op, opIndexes, input, nodes_gpu, 0);
	//op array shared
	simulate(n, totalPlusses, op, opIndexes, input, nodes_gpu, 1);
	//input array shared
	simulate(n, totalPlusses, op, opIndexes, input, nodes_gpu, 2);
	//no arrays shared
	simulate(n, totalPlusses, op, opIndexes, input, nodes_gpu, 3);

	//serial timed test
	simulate(n, totalPlusses, op, opIndexes, input, nodes_normal, 4);

	if (print)
		print_nodes(nodes_normal, totalPlusses);

	//free everything
	hipFree(input);
	for (i = 0; i < totalPlusses; i++)
	{
		free(nodes_gpu[i]);
		free(nodes_normal[i]);
	}
	hipFree(op);
	hipFree(opIndexes);
	hipFree(nodes_gpu);
	hipFree(nodes_normal);

	exit(0);
}
